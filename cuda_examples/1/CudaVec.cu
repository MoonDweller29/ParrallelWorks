#include "CudaVec.h"
#include "hip/hip_runtime.h"

DeviceVec::DeviceVec() : v(NULL)
{}

hipError_t DeviceVec::malloc(size_t size) {
    return hipMalloc((void**)&v, size*sizeof(float));
}


DeviceVec::~DeviceVec() {
    if (v != NULL) {
        hipFree(v);
    }
}

HostVec::HostVec() : v(NULL), _size(0)
{}

HostVec::HostVec(size_t size, bool locked) : v(NULL), _size(size) {
    malloc(size, locked);
}


void HostVec::clear() {
    if (v != NULL) {
        if (_locked) {
            hipFree(v);
        } else {
            delete[] v;
        }
    }
    v = NULL;
}

hipError_t HostVec::malloc(size_t size, bool locked) {
    clear();
    _size = size;
    _locked = locked;
    if (locked) {
        return hipHostMalloc((void**)&v, size*sizeof(float));
    } else {
        v = new float[_size];
        return hipSuccess;
    }
}

float &HostVec::operator[](int i) {
    return v[i];
}

HostVec::~HostVec() {
    clear();
}



PinnedVec::PinnedVec() :
    _host(NULL), _device(NULL), _size(0)
{}

hipError_t PinnedVec::malloc(size_t size) {
    _size = size;
    hipError_t err = hipHostAlloc((void**)&_host, size*sizeof(float), hipHostMallocMapped);
    hipHostGetDevicePointer((void**)&_device, _host, 0);

    return err;
}

float &PinnedVec::operator[](int i) {
    return _host[i];
}

PinnedVec::~PinnedVec() {
    if (_host != NULL) {
        hipFree(_host);
    }
}