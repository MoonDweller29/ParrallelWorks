#include "CudaVec.h"
#include "hip/hip_runtime.h"

DeviceVec::DeviceVec() : v(NULL)
{}

hipError_t DeviceVec::malloc(size_t size) {
    return hipMalloc((void**)&v, size*sizeof(float));
}


DeviceVec::~DeviceVec() {
    if (v != NULL) {
        hipFree(v);
    }
}

HostVec::HostVec() : v(NULL), _size(0)
{}

hipError_t HostVec::malloc(size_t size, bool locked) {
    _size = size;
    _locked = locked;
    if (locked) {
        return hipHostMalloc((void**)&v, size*sizeof(float));
    } else {
        v = new float[_size];
        return hipSuccess;
    }
}

float &HostVec::operator[](int i) {
    return v[i];
}

HostVec::~HostVec() {
    if (v != NULL) {
        if (_locked) {
            hipFree(v);
        } else {
            delete[] v;
        }
    }   
}


// PinnedVec::PinnedVec() :
//     _host(NULL), _device(NULL), _size(NULL)
// {}

// PinnedVec::~PinnedVec() {
//     if (_host != NULL) {
//         hipFree(_host);
//     }
// }