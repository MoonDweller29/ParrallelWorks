#include "CudaVec.h"
#include "hip/hip_runtime.h"

DeviceVec::DeviceVec() : v(NULL)
{}

hipError_t DeviceVec::malloc(size_t size) {
    return hipMalloc((void**)&v, size*sizeof(float));
}


DeviceVec::~DeviceVec() {
    if (v != NULL) {
        hipFree(v);
    }
}

HostVec::HostVec() : v(NULL), _size(0)
{}

hipError_t HostVec::malloc(size_t size, bool locked) {
    _size = size;
    _locked = locked;
    if (locked) {
        return hipHostMalloc((void**)&v, size*sizeof(float));
    } else {
        v = new float[_size];
        return hipSuccess;
    }
}

float &HostVec::operator[](int i) {
    return v[i];
}

HostVec::~HostVec() {
    if (v != NULL) {
        if (_locked) {
            hipFree(v);
        } else {
            delete[] v;
        }
    }   
}



PinnedVec::PinnedVec() :
    _host(NULL), _device(NULL), _size(0)
{}

hipError_t PinnedVec::malloc(size_t size) {
    _size = size;
    hipError_t err = hipHostAlloc((void**)&_host, size*sizeof(float), hipHostMallocMapped);
    hipHostGetDevicePointer((void**)&_device, _host, 0);

    return err;
}

float &PinnedVec::operator[](int i) {
    return _host[i];
}

PinnedVec::~PinnedVec() {
    if (_host != NULL) {
        hipFree(_host);
    }
}