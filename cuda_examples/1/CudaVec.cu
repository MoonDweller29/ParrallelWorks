#include "CudaVec.h"
#include "hip/hip_runtime.h"

DeviceVec::DeviceVec() : v(NULL)
{}

hipError_t DeviceVec::malloc(size_t size) {
	return hipMalloc((void**)&v, size);
}


DeviceVec::~DeviceVec() {
	if (v != NULL) {
    	hipFree(v);
	}
}






// PinnedVec::PinnedVec() {

// }

// PinnedVec::~PinnedVec() {
// 	if (_host != NULL) {
// 		hipFree(_host);
// 	}
// }