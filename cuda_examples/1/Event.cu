#include "Event.h"

Event::Event() {
    hipEventCreate(&event);
}

hipError_t Event::record(hipStream_t stream) {
    return hipEventRecord(event, stream);
}

hipError_t Event::wait(const Event &event) {
    return hipEventSynchronize(*event);
}

Event::~Event() {
    hipEventDestroy(event);
}