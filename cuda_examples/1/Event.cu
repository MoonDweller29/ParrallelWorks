#include "Event.h"

Event::Event() {
    hipEventCreate(&event);
}

hipError_t Event::record(hipStream_t stream) {
    return hipEventRecord(event, stream);
}

hipError_t Event::wait(const Event &event) {
    return hipEventSynchronize(*event);
}

float Event::elapsedTime(const Event &event_1, const Event &event_2) {
    float time;
    wait(event_1);
    wait(event_2);
    hipEventElapsedTime(&time, *event_1, *event_2);

    return time;
}


Event::~Event() {
    hipEventDestroy(event);
}