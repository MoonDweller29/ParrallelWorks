#include "Stream.h"
#include "Event.h"

Stream::Stream() {
	hipStreamCreate(&stream);
}

Stream::~Stream() {
	hipStreamDestroy(stream);
}

hipError_t Stream::wait(const Event& event) const {
	return hipStreamWaitEvent(stream, *event, 0);
} 
