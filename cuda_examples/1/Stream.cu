#include "Stream.h"

Stream::Stream() {
	hipStreamCreate(&stream);
}

Stream::~Stream() {
	hipStreamDestroy(stream);
}