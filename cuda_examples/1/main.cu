#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include "cuda_macro.h"

// gridDim.x - grid size x
// index = blockIdx.x * blockDim.x + threadIdx.x;

// Функция сложения двух векторов
__global__ void addVector(float* left, float* right, float* result)
{
  //Получаем id текущей нити.
  int idx = threadIdx.x;
  
  //Расчитываем результат.
  result[idx] = left[idx] + right[idx];
}


int main(int argc, char const *argv[])
{
    //Выделяем память под вектора
    const int SIZE = 512;
    float* vec1 = new float[SIZE];
    float* vec2 = new float[SIZE];
    float* vec3 = new float[SIZE];

    //Инициализируем значения векторов
    for (int i = 0; i < SIZE; i++)
    {
        vec1[i] = i;
        vec2[i] = i;
    }

    //Указатели на память видеокарты
    float* devVec1;
    float* devVec2;
    float* devVec3;

    //Выделяем память для векторов на видеокарте
    SAFE_CALL( hipMalloc((void**)&devVec1, sizeof(float) * SIZE) )
    SAFE_CALL( hipMalloc((void**)&devVec2, sizeof(float) * SIZE) )
    SAFE_CALL( hipMalloc((void**)&devVec3, sizeof(float) * SIZE) )

    //Копируем данные в память видеокарты
    SAFE_CALL( hipMemcpy(devVec1, vec1, sizeof(float) * SIZE, hipMemcpyHostToDevice) )
    SAFE_CALL( hipMemcpy(devVec2, vec2, sizeof(float) * SIZE, hipMemcpyHostToDevice) )
    
    dim3 gridSize = dim3(1, 1, 1);    //Размер используемого грида
    dim3 blockSize = dim3(SIZE, 1, 1); //Размер используемого блока

    //Выполняем вызов функции ядра
    addVector<<<gridSize, blockSize>>>(devVec1, devVec2, devVec3);
    checkErr();
    // addVector<<<1, SIZE>>>(devVec1, devVec2, devVec3);

    //Хендл event'а
    hipEvent_t syncEvent;

    hipEventCreate(&syncEvent);    //Создаем event
    hipEventRecord(syncEvent, 0);  //Записываем event
    hipEventSynchronize(syncEvent);  //Синхронизируем event

    //Только теперь получаем результат расчета
    SAFE_CALL( hipMemcpy(vec3, devVec3, sizeof(float) * SIZE, hipMemcpyDeviceToHost) )

    for (int i = 0; i < SIZE; i++)
    {
        std::cout<< i <<" : "<< vec3[i] << std::endl;
    }

    hipEventDestroy(syncEvent);

    hipFree(devVec1);
    hipFree(devVec2);
    hipFree(devVec3);

    delete[] vec1;
    delete[] vec2;
    delete[] vec3;

    return 0;
}