#include "CudaVec.h"
#include "hip/hip_runtime.h"
#include <sstream>
#include <iostream>


DeviceVec::DeviceVec() : v(NULL), _size(0)
{}

DeviceVec::DeviceVec(size_t size) : v(NULL), _size(size) {
    malloc(size);
}

hipError_t DeviceVec::malloc(size_t size) {
    clear();
    _size = size;
    return hipMalloc((void**)&v, size*sizeof(double));
}

void DeviceVec::clear() {
    if (v != NULL) {
        hipFree(v);
    }
    v = NULL;
}

DeviceVec::~DeviceVec() {
    clear();
}




HostVec::HostVec() : v(NULL), _size(0)
{}

HostVec::HostVec(size_t size, bool locked) :
    v(NULL), _size(size), _locked(locked)
{
    malloc(size, locked);
}

hipError_t HostVec::malloc(size_t size, bool locked) {
    clear();
    _size = size;
    _locked = locked;
    if (locked) {
        return hipHostMalloc((void**)&v, size*sizeof(double));
    } else {
        v = new double[_size];
        return hipSuccess;
    }
}

double &HostVec::operator[](int i) {
    return v[i];
}

double &HostVec::at(int i) {
    if (i >= 0 && i < _size) {
        return v[i];
    } else {
        std::stringstream s;
        s << "Error in HostVec::at : index " << i << " is out of range [0, " << _size << ")";
        std::cout << s.str() << std::endl;
        throw s.str();
    }
}


void HostVec::clear() {
    if (v != NULL) {
        if (_locked) {
            hipFree(v);
        } else {
            delete[] v;
        }
    }
    v = NULL;
}

HostVec::~HostVec() {
    clear();
}




PinnedVec::PinnedVec() :
    _host(NULL), _device(NULL), _size(0)
{}

hipError_t PinnedVec::malloc(size_t size) {
    _size = size;
    hipError_t err = hipHostAlloc((void**)&_host, size*sizeof(double), hipHostMallocMapped);
    hipHostGetDevicePointer((void**)&_device, _host, 0);

    return err;
}

double &PinnedVec::operator[](int i) {
    return _host[i];
}

PinnedVec::~PinnedVec() {
    if (_host != NULL) {
        hipFree(_host);
    }
}