#include "CudaVec.h"
#include "hip/hip_runtime.h"

DeviceVec::DeviceVec() : v(NULL), _size(0)
{}

DeviceVec::DeviceVec(size_t size) {
    malloc(size);
}

hipError_t DeviceVec::malloc(size_t size) {
    clear();
    _size = size;
    return hipMalloc((void**)&v, size*sizeof(double));
}

void DeviceVec::clear() {
    if (v != NULL) {
        hipFree(v);
    }
    v = NULL;
}

DeviceVec::~DeviceVec() {
    clear();
}




HostVec::HostVec() : v(NULL), _size(0)
{}

HostVec::HostVec(size_t size, bool locked) :
    _size(size), _locked(locked)
{
    malloc(size, locked);
}

hipError_t HostVec::malloc(size_t size, bool locked) {
    clear();
    _size = size;
    _locked = locked;
    if (locked) {
        return hipHostMalloc((void**)&v, size*sizeof(double));
    } else {
        v = new double[_size];
        return hipSuccess;
    }
}

double &HostVec::operator[](int i) {
    return v[i];
}

void HostVec::clear() {
    if (v != NULL) {
        if (_locked) {
            hipFree(v);
        } else {
            delete[] v;
        }
    }
    v = NULL;
}

HostVec::~HostVec() {
    clear();
}




PinnedVec::PinnedVec() :
    _host(NULL), _device(NULL), _size(0)
{}

hipError_t PinnedVec::malloc(size_t size) {
    _size = size;
    hipError_t err = hipHostAlloc((void**)&_host, size*sizeof(double), hipHostMallocMapped);
    hipHostGetDevicePointer((void**)&_device, _host, 0);

    return err;
}

double &PinnedVec::operator[](int i) {
    return _host[i];
}

PinnedVec::~PinnedVec() {
    if (_host != NULL) {
        hipFree(_host);
    }
}